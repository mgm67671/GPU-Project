#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

// CUDA kernel for applying the Sobel filter with shared memory optimization
__global__ void sobelKernelOptimized(const unsigned char* input, unsigned char* output, int width, int height) {
    // Shared memory for the tile
    __shared__ unsigned char sharedMem[18][18]; // Block size (16x16) + 2 for borders

    // Calculate global thread coordinates
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate shared memory coordinates
    int sharedX = threadIdx.x + 1;
    int sharedY = threadIdx.y + 1;

    // Load data into shared memory
    if (x < width && y < height) {
        sharedMem[sharedY][sharedX] = input[y * width + x];

        // Load border pixels into shared memory
        if (threadIdx.x == 0 && x > 0) {
            sharedMem[sharedY][0] = input[y * width + (x - 1)];
        }
        if (threadIdx.x == blockDim.x - 1 && x < width - 1) {
            sharedMem[sharedY][sharedX + 1] = input[y * width + (x + 1)];
        }
        if (threadIdx.y == 0 && y > 0) {
            sharedMem[0][sharedX] = input[(y - 1) * width + x];
        }
        if (threadIdx.y == blockDim.y - 1 && y < height - 1) {
            sharedMem[sharedY + 1][sharedX] = input[(y + 1) * width + x];
        }

        // Load corner pixels
        if (threadIdx.x == 0 && threadIdx.y == 0 && x > 0 && y > 0) {
            sharedMem[0][0] = input[(y - 1) * width + (x - 1)];
        }
        if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0 && x < width - 1 && y > 0) {
            sharedMem[0][sharedX + 1] = input[(y - 1) * width + (x + 1)];
        }
        if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1 && x > 0 && y < height - 1) {
            sharedMem[sharedY + 1][0] = input[(y + 1) * width + (x - 1)];
        }
        if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1 && x < width - 1 && y < height - 1) {
            sharedMem[sharedY + 1][sharedX + 1] = input[(y + 1) * width + (x + 1)];
        }
    }

    // Synchronize threads to ensure shared memory is fully loaded
    __syncthreads();

    // Sobel kernels
    int Gx[3][3] = {
        {-1, 0, 1},
        {-2, 0, 2},
        {-1, 0, 1}
    };

    int Gy[3][3] = {
        {-1, -2, -1},
        { 0,  0,  0},
        { 1,  2,  1}
    };

    // Apply Sobel filter if within bounds
    if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
        int sumX = 0;
        int sumY = 0;

        // Convolution with Sobel kernels
        for (int ky = -1; ky <= 1; ++ky) {
            for (int kx = -1; kx <= 1; ++kx) {
                int pixel = sharedMem[sharedY + ky][sharedX + kx];
                sumX += pixel * Gx[ky + 1][kx + 1];
                sumY += pixel * Gy[ky + 1][kx + 1];
            }
        }

        // Compute gradient magnitude
        int magnitude = sqrtf(sumX * sumX + sumY * sumY);
        magnitude = min(255, magnitude); // Clamp to 255

        // Write the result to the output image
        output[y * width + x] = static_cast<unsigned char>(magnitude);
    }
}

int main() {
    // Load the input image in grayscale
    cv::Mat input = cv::imread("input_image.jpg", cv::IMREAD_GRAYSCALE);
    if (input.empty()) {
        std::cerr << "Error: Could not load input image!" << std::endl;
        return -1;
    }

    int width = input.cols;
    int height = input.rows;

    // Allocate memory for the output image
    cv::Mat output(height, width, CV_8UC1);

    // Allocate device memory
    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, width * height * sizeof(unsigned char));
    hipMalloc(&d_output, width * height * sizeof(unsigned char));

    // Copy input image to device
    hipMemcpy(d_input, input.data, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Launch the optimized Sobel kernel
    sobelKernelOptimized<<<gridSize, blockSize>>>(d_input, d_output, width, height);

    // Copy the result back to the host
    hipMemcpy(output.data, d_output, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Save the result
    cv::imwrite("gpu_output_image_optimized.jpg", output);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}